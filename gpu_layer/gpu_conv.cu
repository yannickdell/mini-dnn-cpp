#include "hip/hip_runtime.h"
#include "../../src/layer/conv.h"

#define TILE_WIDTH 32

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}
// v1_use global memory
__global__ void matMulKernel_v1(float* A, float* B, float* C, int M, int N, int P) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < M && col < P) {
    float sum = 0;
    for (int i = 0; i < N; i++) {
      sum += A[row * N + i] * B[i * P + col];
    }
    C[row * P + col] = sum;
  }
}

// v2_use shared memory
__global__ void matMulKernel_v2(float* A, float* B, float* C, int M, int N, int P) {
  // Allocate shared memory
  __shared__ float As[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float sum = 0;

  // Loop over the tiles of the input matrices
  for (int m = 0; m < ((N - 1)/TILE_WIDTH + 1); ++m) {

    // Load one tile of A and B into shared memory
    if (row < M && m*TILE_WIDTH + threadIdx.x < N)
      As[threadIdx.y][threadIdx.x] = A[row*N + m*TILE_WIDTH + threadIdx.x];
    else
      As[threadIdx.y][threadIdx.x] = 0.0;

    if (m*TILE_WIDTH + threadIdx.y < N && col < P)
      Bs[threadIdx.y][threadIdx.x] = B[(m*TILE_WIDTH + threadIdx.y)*P + col];
    else
      Bs[threadIdx.y][threadIdx.x] = 0.0;

    // Synchronize to make sure the tile is loaded
    __syncthreads();

    // Multiply the elements of the tile and accumulate the results
    for (int k = 0; k < TILE_WIDTH; ++k)
      sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];

    // Synchronize to make sure the computation is done before loading the next tile
    __syncthreads();
  }

  // Write the computed value to the output matrix
  if (row < M && col < P)
    C[row * P + col] = sum;
}

// __global__ void im2colKernel(const float* image, float* data_col, int height_in, int width_in, int height_out, int width_out, int height_kernel, int width_kernel, int stride, int pad_h, int pad_w) {
//   int i = blockIdx.x * blockDim.x + threadIdx.x;

//   if (i < height_out * width_out) {
//     int step_h = i / width_out;
//     int step_w = i % width_out;
//     int start_idx = step_h * width_in * stride + step_w * stride;  // left-top idx of window

//     for (int j = 0; j < height_kernel * width_kernel; j ++) {
//       int cur_col = start_idx % width_in + j % width_kernel - pad_w;  // col after padding
//       int cur_row = start_idx / width_in + j / width_kernel - pad_h;

//       if (cur_col < 0 || cur_col >= width_in || cur_row < 0 || cur_row >= height_in) {
//         data_col[i * height_kernel * width_kernel + j] = 0;
//       }
//       else {
//         int pick_idx = cur_row * width_in + cur_col;
//         data_col[i * height_kernel * width_kernel + j] = image[pick_idx];  // pick which pixel
//       }
//     }
//   }
// }

// void Conv::im2col(const Vector& image, Matrix& data_col) {
//   // Convert Eigen Vector to array
//   const float* image_array = image.data();

//   // Calculate dimensions
//   int hw_in = height_in * width_in;
//   int hw_kernel = height_kernel * width_kernel;
//   int hw_out = height_out * width_out;

//   // Allocate GPU memory
//   float* d_image;
//   float* d_data_col;
//   hipMalloc(&d_image, hw_in * sizeof(float));
//   hipMalloc(&d_data_col, hw_out * hw_kernel * sizeof(float));

//   // Copy data to GPU
//   hipMemcpy(d_image, image_array, hw_in * sizeof(float), hipMemcpyHostToDevice);

//   // Call GPU kernel
//   int numThreads = 256;
//   int numBlocks = (hw_out + numThreads - 1) / numThreads;

//   im2colKernel<<<numBlocks, numThreads,smem_size>>>(d_image, d_data_col, height_in, width_in, height_out, width_out, height_kernel, width_kernel, stride, pad_h, pad_w);
//   hipDeviceSynchronize();

//   // Copy result back to CPU
//   float* data_col_array = new float[hw_out * hw_kernel];
//   hipMemcpy(data_col_array, d_data_col, hw_out * hw_kernel * sizeof(float), hipMemcpyDeviceToHost);

//   // Convert result array back to Eigen matrix
//   data_col = Eigen::Map<Matrix>(data_col_array, hw_out, hw_kernel);

//   // Free GPU memory
//   hipFree(d_image);
//   hipFree(d_data_col);

//   // Delete result array
//   delete[] data_col_array;
// }

// v3_ use stream
void Conv::forward(const Matrix& bottom){
  int n_sample = bottom.cols();
  top.resize(height_out * width_out * channel_out, n_sample);
  data_cols.resize(n_sample);

  hipStream_t stream1, stream2;
  CHECK(hipStreamCreate(&stream1));
  CHECK(hipStreamCreate(&stream2));

  for (int i = 0; i < n_sample; i ++) {
    // im2col
    Matrix data_col;
    im2col(bottom.col(i), data_col);
    data_cols[i] = data_col;

    float* data_col_array = data_col.data();
    float* weight_array = weight.data();
    float* result_array = new float[data_col.rows() * weight.cols()];

    // Allocate GPU memory
    float* d_data_col;
    float* d_weight;
    float* d_result;
    CHECK(hipMalloc(&d_data_col, data_col.size() * sizeof(float)));
    CHECK(hipMalloc(&d_weight, weight.size() * sizeof(float)));
    CHECK(hipMalloc(&d_result, data_col.rows() * weight.cols() * sizeof(float)));

    // Copy data to GPU
    CHECK(hipMemcpyAsync(d_data_col, data_col_array, data_col.size() * sizeof(float), hipMemcpyHostToDevice, stream1));
    CHECK(hipMemcpyAsync(d_weight, weight_array, weight.size() * sizeof(float), hipMemcpyHostToDevice, stream2));

    // Call GPU kernel
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((weight.cols() + threadsPerBlock.x - 1) / threadsPerBlock.x, (data_col.rows() + threadsPerBlock.y - 1) / threadsPerBlock.y);
    int smem_size = (threadsPerBlock.x + TILE_WIDTH - 1)*(threadsPerBlock.y + TILE_WIDTH - 1) * sizeof(float);
    matMulKernel_v2<<<numBlocks, threadsPerBlock, smem_size>>>(d_data_col, d_weight, d_result, data_col.rows(), data_col.cols(), weight.cols());
    hipDeviceSynchronize();

    // Copy result back to CPU
    CHECK(hipMemcpy(result_array, d_result, data_col.rows() * weight.cols() * sizeof(float), hipMemcpyDeviceToHost));

    // Convert result array back to Eigen matrix
    Matrix result = Eigen::Map<Matrix>(result_array, data_col.rows(), weight.cols());

    result.rowwise() += bias.transpose();
    top.col(i) = Eigen::Map<Vector>(result.data(), result.size());

    // Free GPU memory
    CHECK(hipFree(d_data_col));
    CHECK(hipFree(d_weight));
    CHECK(hipFree(d_result));

    // Delete result array
    delete[] result_array;
  }

  CHECK(hipStreamDestroy(stream1));
  CHECK(hipStreamDestroy(stream2));
}